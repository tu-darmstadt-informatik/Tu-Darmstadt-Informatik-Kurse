#include "hip/hip_runtime.h"
//
//  kernel.cu
//
//  Created by Arya Mazaheri on 01/12/2018.
//

#include <iostream>
#include <algorithm>
#include <cmath>
#include "ppm.h"

using namespace std;

/*********** Gray Scale Filter  *********/

/**
 * Converts a given 24bpp image into 8bpp grayscale using the GPU.
 */
__global__
void cuda_grayscale(int width, int height, BYTE *image, BYTE *image_out) //Kernel
{
    //TODO (9 pt): implement grayscale filter kernel AUFGABE 4
		  int idx = threadIdx.x + blockIdx.x * blockDim.x;
}


// 1D Gaussian kernel array values of a fixed size (make sure the number > filter size d)
//TODO: Define the cGaussian array on the constant memory (2 pt)

void cuda_updateGaussian(int r, double sd)
{
	float fGaussian[64];
	for (int i = 0; i < 2*r +1 ; i++)
	{
		float x = i - r;
		fGaussian[i] = expf(-(x*x) / (2 * sd*sd));
	}
	//TODO: Copy computed fGaussian to the cGaussian on device memory (2 pts)
	hipMemcpyToSymbol(HIP_SYMBOL(cGaussian), /* TODO */);
}

//TODO: implement cuda_gaussian() kernel (3 pts)


/*********** Bilateral Filter  *********/
// Parallel (GPU) Bilateral filter kernel
__global__ void cuda_bilateral_filter(BYTE* input, BYTE* output,
	int width, int height,
	int r, double sI, double sS)
{
	//TODO: implement bilateral filter kernel (9 pts)
}


void gpu_pipeline(const Image & input, Image & output, int r, double sI, double sS)
{
	// Events to calculate gpu run time
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	// GPU related variables
	BYTE *d_input = NULL;
	BYTE *d_image_out[2] = {0}; //temporary output buffers on gpu device
	int image_size = input.cols*input.rows;
	int suggested_blockSize;   // The launch configurator returned block size
	int suggested_minGridSize; // The minimum grid size needed to achieve the maximum occupancy for a full device launch

	// ******* Grayscale kernel launch *************

	//Creating the block size for grayscaling kernel
	hipOccupancyMaxPotentialBlockSize( &suggested_minGridSize, &suggested_blockSize, cuda_grayscale);
//Jeder Thread soll einen Pixelbearbeiten
        int block_dim_x, block_dim_y;
        block_dim_x = block_dim_y = (int) sqrt(suggested_blockSize);

    //    dim3 gray_block(/* TODO */suggested_blockSize, TODOoccupancy ); // 2 pts
				dim3 gray_block(/* TODO */block_dim_x, block_dim_y ); // 2 pts
				//TODO: Calculate grid size to cover the whole image - 2 pts
				//dim3 gray_grid(input.cols / block_dim_x , input.rows / block_dim_y);
				dim3 gray_grid((input.cols+ suggested_blockSize -1) /suggested_blockSize, (input.rows+ suggested_blockSize -1) /suggested_blockSize);
				int AnzahlderThreadsproblock = gray_grid.x * gray_grid.y; //Beispiel ka welchen man nehmen soll //Variable zum besseren Verständniss
        //Funktioniert nur für quadrat zahleń int gridsize = (image_size+ suggested_blockSize -1) /suggested_blockSize;
				//anzahl der Threads im Block könnten ja prizipiell eine ganze Zeile abdecken



        // Allocate the intermediate image buffers for each step
        Image img_out(input.cols, input.rows, 1, "P5");
				BYTE *puffer[2];
				int InitByte = 0;
				size_t gesamtgroeße = i; //TODO wert reinschreiben
				int anzahlbytes; //TODO wert reinschreiben
        for (int i = 0; i < 2; i++)
        {
            //TODO: allocate memory on the device (2 pts)
						hipMalloc((void**) &puffer[i], (sizeof(BYTE)) * image_size);
            //TODO: intialize allocated memory on device to zero (2 pts)
						hipMemset(&puffer[i], InitByte, image_size);
        }

        //copy input image to device
        //TODO: Allocate memory on device for input image (2 pts)
				size_t gesamtgrößeBildpuffer;
				BYTE *pufferEingabebild;

				//hipMalloc((void**) &pufferEingabebild, sizeof(BYTE) * image_size);
				hipMalloc((void**) &pufferEingabebild, 3* image_size); // r g b
				//evtl doch das		 hipMemset2D ( &pufferEingabebild, size_t pitch, int  value, size_t width, size_t height ) //aus den duration_cast
//TODO: Copy input image into the device memory (2 pts)
				Image img = readPPM("test_input.ppm");
				cudamemcpy(&pufferEingabebild, &input.pixels /** holt den Bytearr welcher alle werte vom img beinhaltet */, image_size * (sizeof(int)), hipMemcpyHostToDevice);

        hipEventRecord(start, 0); // start timer
        // Convert input image to grayscale
        //TODO: Launch cuda_grayscale() (2 pts)
					cuda_grayscale<<< gray_size, gray_block>>>( input.cols, input.rows, &input.pixels,&pufferEingabebild ); //Das sind die Parameter der grayscale methode//Folie 69

        hipEventRecord(stop, 0); // stop timer
        hipEventSynchronize(stop);

        // Calculate and print kernel run time
	hipEventElapsedTime(&time, start, stop);
	cout << "GPU Grayscaling time: " << time << " (ms)\n";
	cout << "Launched blocks of size " << gray_block.x * gray_block.y << endl; //ist sowas wie ein printf befehl denke ich

        //TODO: transfer image from device to the main memory for saving onto the disk (2 pts)
				cudamemcpy(img_out, &pufferEingabebild,  image_size * (sizeof(int)), hipMemcpyDeviceToHost);
		    savePPM(img_out, "image_gpu_gray.ppm");


	// ******* Bilateral filter kernel launch *************

	//Creating the block size for grayscaling kernel
	hipOccupancyMaxPotentialBlockSize( &suggested_minGridSize, &suggested_blockSize, cuda_bilateral_filter); //cuda_bil.. ist kernel

        block_dim_x = block_dim_y = (int) sqrt(suggested_blockSize);

        dim3 bilateral_block(/* TODO */ suggested_blockSize, TODOoccupancy); // 2 pts steht im Internetz so

        //TODO: Calculate grid size to cover the whole image - 2pts

        // Create gaussain 1d array
        cuda_updateGaussian(r,sS);

        hipEventRecord(start, 0); // start timer
	//TODO: Launch cuda_bilateral_filter() (2 pts)
        hipEventRecord(stop, 0); // stop timer
        hipEventSynchronize(stop);

        // Calculate and print kernel run time
        hipEventElapsedTime(&time, start, stop);
        cout << "GPU Bilateral Filter time: " << time << " (ms)\n";
        cout << "Launched blocks of size " << bilateral_block.x * bilateral_block.y << endl;

        // Copy output from device to host
	//TODO: transfer image from device to the main memory for saving onto the disk (2 pts)


        // ************** Finalization, cleaning up ************

        // Free GPU variables
	//TODO: Free device allocated memory (3 pts)
}
