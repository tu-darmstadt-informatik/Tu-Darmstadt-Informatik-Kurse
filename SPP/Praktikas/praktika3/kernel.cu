#include "hip/hip_runtime.h"
//
//  kernel.cu
//
//  Created by Arya Mazaheri on 01/12/2018.
//

#include <iostream>
#include <algorithm>
#include <cmath>
#include "ppm.h"

using namespace std;

/*********** Gray Scale Filter  *********/

/**
 * Converts a given 24bpp image into 8bpp grayscale using the GPU.
 */
__global__
void cuda_grayscale(int width, int height, BYTE *image, BYTE *image_out)
{
    //TODO (9 pt): implement grayscale filter kernel
		int bx = blockIdx.x;
		int by = blockIdx.y;





		for (int h = 0; h < height; h++)
		{
				int offset_out = h * width;      // 1 color per pixel
				int offset     = offset_out * 3; // 3 colors per pixel

				for (int w = 0; w < width; w++)
				{
						BYTE *pixel = &image[offset + w * 3];

						// Convert to grayscale following the "luminance" model
						image_out[offset_out + w] = pixel[0] * 0.0722f + // B
						pixel[1] * 0.7152f + // G
						pixel[2] * 0.2126f;  // R
				}
		}
}


// 1D Gaussian kernel array values of a fixed size (make sure the number > filter size d)
//TODO: Define the cGaussian array on the constant memory (2 pt)
__constant__
float cGaussian[64];

void cuda_updateGaussian(int r, double sd)
{
	float fGaussian[64];
	for (int i = 0; i < 2*r +1 ; i++)
	{
		float x = i - r;
		fGaussian[i] = expf(-(x*x) / (2 * sd*sd));
	}
	//TODO: Copy computed fGaussian to the cGaussian on device memory (2 pts)
	hipMemcpyToSymbol(HIP_SYMBOL(&cGaussian), &fGaussian, 0, hipMemcpyDeviceToDevice);
}

//TODO: implement cuda_gaussian() kernel (3 pts)
__device__
void cuda_gaussian(float x, double sigma, float a, float b, double* out)
{
	double output = a * b *	expf(-(powf(x, 2)) / (2 * powf(sigma, 2)));
	out = &output;
}


/*********** Bilateral Filter  *********/
// Parallel (GPU) Bilateral filter kernel
__global__ void cuda_bilateral_filter(BYTE* input, BYTE* output,
	int width, int height,
	int r, double sI, double sS)
	{
		for(int h = 0; h < height; h++){
			for(int w = 0; w < width; w++){
				double iFiltered = 0;
				double wP = 0;
				// Get the centre pixel value
				unsigned char centrePx = input[h*width+w];
				// Iterate through filter size from centre pixel
				for (int dy = -r; dy <= r; dy++) {
					int neighborY = h+dy;
					if (neighborY < 0)
	                    neighborY = 0;
	                else if (neighborY >= height)
	                    neighborY = height - 1;
					for (int dx = -r; dx <= r; dx++) {
						int neighborX = w+dx;
						if (neighborX < 0)
		                    neighborX = 0;
		                else if (neighborX >= width)
		                    neighborX = width - 1;
						// Get the current pixel; value
						unsigned char currPx = input[neighborY*width+neighborX];
						// Weight = 1D Gaussian(x_axis) * 1D Gaussian(y_axis) * Gaussian(Range or Intensity difference)
						//TODO: implement bilateral filter kernel (9 pts)
						double w;
						cuda_gaussian(centrePx - currPx, sI, cGaussian[dy + r], cGaussian[dx + r], &w);

						iFiltered += w * currPx;
						wP += w;
					}
				}
				output[h*width + w] = iFiltered / wP;
			}
		}
	}

	//<<<bilateral_grid, bilateral_block>>>


void gpu_pipeline(const Image & input, Image & output, int r, double sI, double sS)
{
	// Events to calculate gpu run time
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// GPU related variables
	BYTE *d_input = NULL;
	BYTE *d_image_out[2] = {0}; //temporary output buffers on gpu device
	int image_size = input.cols*input.rows;
	int suggested_blockSize;   // The launch configurator returned block size
	int suggested_minGridSize; // The minimum grid size needed to achieve the maximum occupancy for a full device launch

	// ******* Grayscale kernel launch *************

	//Creating the block size for grayscaling kernel
	hipOccupancyMaxPotentialBlockSize( &suggested_minGridSize, &suggested_blockSize, cuda_grayscale);

        int block_dim_x, block_dim_y;
        block_dim_x = block_dim_y = (int) sqrt(suggested_blockSize);

        dim3 gray_block(block_dim_x, block_dim_y); // 2 pts

				int grid_x, grid_y;
				grid_x = (input.cols + block_dim_x - 1) / block_dim_x;
				grid_y = (input.rows + block_dim_x - 1) / block_dim_y;


        //TODO: Calculate grid size to cover the whole image - 2 ptsd
				dim3 gray_grid(grid_x, grid_y);
				printf("HIII\n");

        // Allocate the intermediate image buffers for each step
        Image img_out(input.cols, input.rows, 1, "P5");
        for (int i = 0; i < 2; i++)
        {
            //TODO: allocate memory on the device (2 pts)
						hipMalloc((void**) &d_image_out[i], sizeof(BYTE) * image_size);
            //TODO: intialize allocated memory on device to zero (2 pts)
						hipMemset(/*(void**)*/ &d_image_out[i], 0, sizeof(BYTE) * image_size);
        }

        //copy input image to device
        //TODO: Allocate memory on device for input image (2 pts)
				hipMalloc((void**) &d_input, sizeof(BYTE) * image_size * 3);
        //TODO: Copy input image into the device memory         dim3 gray_block(block_dim_x, block_dim_y); // 2 pts
				hipMemcpy(d_input, input.pixels, image_size * 3, hipMemcpyHostToDevice);



        hipEventRecord(start, 0); // start timer
        // Convert input image to grayscale
        //TODO: Launch cuda_grayscale() (2 pts)
				cuda_grayscale<<<gray_grid, gray_block>>>(input.cols, input.rows, d_input, d_image_out[0]);
				cuda_grayscale<<<gray_grid, gray_block>>>(input.cols, input.rows, d_input, d_image_out[1]);


        hipEventRecord(stop, 0); // stop timer
        hipEventSynchronize(stop);

        // Calculate and print kernel run time
	hipEventElapsedTime(&time, start, stop);
	cout << "GPU Grayscaling time: " << time << " (ms)\n";
	cout << "Launched blocks of size " << gray_block.x * gray_block.y << endl;

        //TODO: transfer image from device to the main memory for saving onto the disk (2 pts)
				hipMemcpy(output.pixels, d_image_out[0], image_size, hipMemcpyDeviceToHost);
        savePPM(output, "image_gpu_gray.ppm");


	// ******* Bilateral filter kernel launch *************

	//Creating the block size for grayscaling kernel
	hipOccupancyMaxPotentialBlockSize( &suggested_minGridSize, &suggested_blockSize, cuda_bilateral_filter);

        block_dim_x = block_dim_y = (int) sqrt(suggested_blockSize);

        dim3 bilateral_block(block_dim_x, block_dim_y/* TODO */); // 2 pts

        //TODO: Calculate grid size to cover the whole image - 2pts
				dim3 bilateral_grid((input.cols + block_dim_x - 1)/block_dim_x, (input.rows + block_dim_x - 1)/block_dim_y);

        // Create gaussain 1d array
        cuda_updateGaussian(r,sS);

        hipEventRecord(start, 0); // start timer
				//TODO: Launch cuda_bilateral_filter() (2 pts)
				cuda_bilateral_filter<<<bilateral_grid, bilateral_block>>>(input.pixels, output.pixels, input.cols, input.rows, r, sI, sS);

				//cpu_bilateral_filter(BYTE* input, BYTE* output, int width, int height, int r, double sI, double sS)

        hipEventRecord(stop, 0); // stop timer
        hipEventSynchronize(stop);

        // Calculate and print kernel run time
        hipEventElapsedTime(&time, start, stop);
        cout << "GPU Bilateral Filter time: " << time << " (ms)\n";
        cout << "Launched blocks of size " << bilateral_block.x * bilateral_block.y << endl;

        // Copy output from device to host
	//TODO: transfer image from device to the main memory for saving onto the disk (2 pts)


        // ************** Finalization, cleaning up ************

        // Free GPU variables
	//TODO: Free device allocated memory (3 pts)
}
