#include "hip/hip_runtime.h"
//
//  kernel.cu
//
//  Created by Arya Mazaheri on 01/12/2018.
//

#include <iostream>
#include <algorithm>
#include <cmath>
#include "ppm.h"

using namespace std;

/*********** Gray Scale Filter  *********/

/**
 * Converts a given 24bpp image into 8bpp grayscale using the GPU.
 */
__global__
void cuda_grayscale(int width, int height, BYTE *image, BYTE *image_out)
{
    //TODO (9 pt): implement grayscale filter kernel
}


// 1D Gaussian kernel array values of a fixed size (make sure the number > filter size d)
//TODO: Define the cGaussian array on the constant memory (2 pt)

void cuda_updateGaussian(int r, double sd)
{
	float fGaussian[64];
	for (int i = 0; i < 2*r +1 ; i++)
	{
		float x = i - r;
		fGaussian[i] = expf(-(x*x) / (2 * sd*sd));
	}
	//TODO: Copy computed fGaussian to the cGaussian on device memory (2 pts)
	hipMemcpyToSymbol(HIP_SYMBOL(cGaussian), /* TODO */);
}

//TODO: implement cuda_gaussian() kernel (3 pts)


/*********** Bilateral Filter  *********/
// Parallel (GPU) Bilateral filter kernel
__global__ void cuda_bilateral_filter(BYTE* input, BYTE* output,
	int width, int height,
	int r, double sI, double sS)
{
	//TODO: implement bilateral filter kernel (9 pts)
}


void gpu_pipeline(const Image & input, Image & output, int r, double sI, double sS)
{
	// Events to calculate gpu run time
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// GPU related variables
	BYTE *d_input = NULL;
	BYTE *d_image_out[2] = {0}; //temporary output buffers on gpu device
	int image_size = input.cols*input.rows;
	int suggested_blockSize;   // The launch configurator returned block size 
	int suggested_minGridSize; // The minimum grid size needed to achieve the maximum occupancy for a full device launch

	// ******* Grayscale kernel launch *************

	//Creating the block size for grayscaling kernel
	hipOccupancyMaxPotentialBlockSize( &suggested_minGridSize, &suggested_blockSize, cuda_grayscale);
        
        int block_dim_x, block_dim_y;
        block_dim_x = block_dim_y = (int) sqrt(suggested_blockSize); 

        dim3 gray_block(/* TODO */); // 2 pts

        //TODO: Calculate grid size to cover the whole image - 2 pts

        // Allocate the intermediate image buffers for each step
        Image img_out(input.cols, input.rows, 1, "P5");
        for (int i = 0; i < 2; i++)
        {  
            //TODO: allocate memory on the device (2 pts)
            //TODO: intialize allocated memory on device to zero (2 pts)
        }

        //copy input image to device
        //TODO: Allocate memory on device for input image (2 pts)
        //TODO: Copy input image into the device memory (2 pts)

        hipEventRecord(start, 0); // start timer
        // Convert input image to grayscale
        //TODO: Launch cuda_grayscale() (2 pts)
        hipEventRecord(stop, 0); // stop timer
        hipEventSynchronize(stop);

        // Calculate and print kernel run time
	hipEventElapsedTime(&time, start, stop);
	cout << "GPU Grayscaling time: " << time << " (ms)\n";
	cout << "Launched blocks of size " << gray_block.x * gray_block.y << endl;
    
        //TODO: transfer image from device to the main memory for saving onto the disk (2 pts)
        savePPM(img_out, "image_gpu_gray.ppm");
	

	// ******* Bilateral filter kernel launch *************
	
	//Creating the block size for grayscaling kernel
	hipOccupancyMaxPotentialBlockSize( &suggested_minGridSize, &suggested_blockSize, cuda_bilateral_filter); 
        
        block_dim_x = block_dim_y = (int) sqrt(suggested_blockSize); 

        dim3 bilateral_block(/* TODO */); // 2 pts

        //TODO: Calculate grid size to cover the whole image - 2pts

        // Create gaussain 1d array
        cuda_updateGaussian(r,sS);

        hipEventRecord(start, 0); // start timer
	//TODO: Launch cuda_bilateral_filter() (2 pts)
        hipEventRecord(stop, 0); // stop timer
        hipEventSynchronize(stop);

        // Calculate and print kernel run time
        hipEventElapsedTime(&time, start, stop);
        cout << "GPU Bilateral Filter time: " << time << " (ms)\n";
        cout << "Launched blocks of size " << bilateral_block.x * bilateral_block.y << endl;

        // Copy output from device to host
	//TODO: transfer image from device to the main memory for saving onto the disk (2 pts)


        // ************** Finalization, cleaning up ************

        // Free GPU variables
	//TODO: Free device allocated memory (3 pts)
}
